#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


#define N_THREADS 1024

// A struct is used to keep function prototypes concise
typedef struct CLIENT_PARAMS {
  float   t; // t        <- current time
  int     m; // m        <- number of devices in the cluster
  float*  B; // B[k]     <- available bandwidth at kth device
  float*  C; // C[k]     <- shared processing capacity of the kth device
  float*  L; // C[k]     <- latency between the synchronizer and the kth
             //             device
  int*    Q; // Q[k]     <- is there a task in queue for the kth device
  float*  E; // E[k]     <- the execution load of the next task for the 
             //             kth device
  float*  I; // I[k]     <- size of the transferable input data and
             //             executable code of the next task for the kth
             //             device
  float*  R; // R[k]     <- size of the results of the next task for the 
             //             kth device
  float* Tc; // Tc[k]    -> completion time of current/last task at kth
             //             device
  int*    K; // K[k]     -> number of tasks loaded by kth device (used by
             //             synchronizer to determine next task)
} CLIENT_PARAMS;

void alloc_client_params(int m, CLIENT_PARAMS* cp) {
  cp->t = 0;
  cp->m = m;
  hipMallocManaged(&cp->B,  m * sizeof(float));
  hipMallocManaged(&cp->C,  m * sizeof(float));
  hipMallocManaged(&cp->L,  m * sizeof(float));
  hipMallocManaged(&cp->Q,  m * sizeof(int));
  hipMallocManaged(&cp->E,  m * sizeof(float));
  hipMallocManaged(&cp->I,  m * sizeof(float));
  hipMallocManaged(&cp->R,  m * sizeof(float));
  hipMallocManaged(&cp->Tc, m * sizeof(float));
  hipMallocManaged(&cp->K,  m * sizeof(int));
}

void init_client_params_zeros(CLIENT_PARAMS* cp) {
  int i;
  cp->t = 0;

  for(i = 0; i < cp->m; i++) {
    cp->B[i]  = 0;
    cp->C[i]  = 0;
    cp->L[i]  = 0;
    cp->Q[i]  = 0;
    cp->E[i]  = 0;
    cp->I[i]  = 0;
    cp->R[i]  = 0;
    cp->Tc[i] = 0;
    cp->K[i]  = 0;
  }
}

void free_client_params(CLIENT_PARAMS* cp) {
  hipFree(cp->B);
  hipFree(cp->C);
  hipFree(cp->L);
  hipFree(cp->Q);
  hipFree(cp->E);
  hipFree(cp->I);
  hipFree(cp->R);
  hipFree(cp->Tc);
  hipFree(cp->K);
}

// Simulates the execution of a task on the cluster by computing the completion
// time for the task performed. Assumes bandwidth is only limited by
// synchronizer and that latency is constant. At most one task is queued up for
// when the node becomes free. When t > Tc[k], the new task is loaded, and the
// appropriate variables changed.
__global__ void client_kernel(CLIENT_PARAMS cp) {
  // See CLIENT_PARAMS struct definition for explanation of variables below
  const float  t = cp.t;
  const int    m = cp.m;
  const float* B = cp.B; 
  const float* C = cp.C; // might not be used in the client kernel, may only
                         // be needed for synchronizer (?)
  const float* L = cp.L; 
  const int*   Q = cp.Q;
  const float* E = cp.E; // ditto
  const float* I = cp.I;
  const float* R = cp.R; 
  
  float* Tc = cp.Tc;
  int*    K = cp.K;

  unsigned int k = blockIdx.x*blockDim.x + threadIdx.x;

  float t_exec;

  while(k < m) {
    // Compute time to execute task (might not be used)
    t_exec = L[k] + I[k] * B[k] + I[k] * R[k]; // + computation time (?)
          //  ^          ^             ^
          // Latency     ^       upload results
          //      download task
      
    // Add this execution time to the next completioan time, if applicable
    K[k] += (t <= Tc[k] && Q[k]);
    Tc[k] += (t <= Tc[k] && Q[k]) * t_exec;

    k += blockDim.x * gridDim.x;
  }
}

typedef struct SYNCZR_PARAMS {

} SYNCZR_PARAMS;

int main() {
  CLIENT_PARAMS cp;
  alloc_client_params(1048576, &cp);
  init_client_params_zeros(&cp);

  client_kernel<<<(cp.m + N_THREADS - 1) / N_THREADS, N_THREADS>>>(cp);

  free_client_params(&cp);

  printf("Done!\n");
  return 0;
}
