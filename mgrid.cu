#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "client.h"

#define N_THREADS 1024


int main() {
  CLIENT_PARAMS cp;
  alloc_client_params(1048576, &cp);
  init_client_params_zeros(&cp);

  client_step_kernel<<<(cp.m + N_THREADS - 1) / N_THREADS, N_THREADS>>>(cp);

  free_client_params(&cp);

  printf("Done!\n");
  return 0;
}
